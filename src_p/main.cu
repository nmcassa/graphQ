#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

extern "C" {
#include "sparse.h"
}

#define NUM_ITER 1
#define STRIDE 2

__device__ void num_in_row(int* rows, int* vals, int row, int index, int* count, int size);

__global__ void fun(int* row, int* col, int* val, int size) {
	int index = threadIdx.x * STRIDE;
	for (int i = 0; i < NUM_ITER; i++) {
		int* curr_edge = (int *) malloc(sizeof(int) * 3);

		curr_edge[0] = row[index];
		curr_edge[1] = col[index];
		curr_edge[2] = val[index];

		int num_row = 0;
		num_in_row(row, val, curr_edge[0], threadIdx.x * STRIDE, &num_row, size);
		printf("%d: %d %d %d\n", num_row, curr_edge[0], curr_edge[1], curr_edge[2]);
	}
}

int main(void) {
	int nodes = 9;
	int edges = 15;

	struct Triplet T = read_triplet("../data/nick/clean.csv", nodes, nodes, edges);

	int count = 0;

	int *row, *col, *val;

	hipMalloc(&row, edges * sizeof(int));
	hipMalloc(&col, edges * sizeof(int));
	hipMalloc(&val, edges * sizeof(int));

	hipMemcpy(row, T.row, edges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(col, T.col, edges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(val, T.val, edges * sizeof(int), hipMemcpyHostToDevice);

	int threads = nodes / STRIDE;
	fun<<<1, threads>>>(row, col, val, edges);
	hipDeviceSynchronize();

	printf("\n\n%d\n", count);

	return 0;
}

__device__ void num_in_row(int* rows, int* vals, int row, int index, int* count, int size) {
	for (int i = index; i < size; i++) {
		if (rows[i] == row && vals[i] == 1) { //unexplored
			(*count)++;
		} else {
			break;
		}
	}
}


