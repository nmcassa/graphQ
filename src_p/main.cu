#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

extern "C" {
#include "sparse.h"
}

#define NUM_ITER 2
#define STRIDE 2

__device__ void find_row_index(int* rows, int col, int size, int* ret);

__global__ void fun(int* row, int* col, int* val, int size) {
	int index = threadIdx.x;

	for (int i = row[index]; i < row[index + 1]; i++) { //go through each row
		for (int j = row[col[i]]; j < row[col[i] + 1]; j++) { //go to each column in that row
			if (val[j] > val[i]) { //check if the value held is larger
				val[j] = val[i];
			}
		}
	}
}

int main(void) {
	int nodes = 9;
	int edges = 15;

	struct Triplet T = read_triplet("../data/nick/clean.csv", nodes, nodes, edges);

	for (int i = 0; i < edges; i++) { //set all values to be their row
		T.val[i] = T.row[i];
		printf("[%d, %d]: %d\n", T.row[i], T.col[i], T.val[i]);
	}

	struct CSR C = Triplet_to_CSR(&T);
	destroy_Triplet(&T);

	int *row, *col, *val;

	hipMalloc(&row, edges * sizeof(int));
	hipMalloc(&col, edges * sizeof(int));
	hipMalloc(&val, edges * sizeof(int));

	hipMemcpy(row, C.rowPtr, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(col, C.colIndex, edges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(val, C.val, edges * sizeof(int), hipMemcpyHostToDevice);

	//int threads = nodes;
	int threads = nodes;
	for (int i = 0; i < NUM_ITER; i++) {
		fun<<<1, threads>>>(row, col, val, edges);
		hipDeviceSynchronize();
	}

	hipMemcpy(C.val, val, edges * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < edges; i++) {
		printf("%d ", C.val[i]);
	}
	printf("\n");
	
	return 0;
}

__device__ void find_row_index(int* rows, int col, int size, int* ret) {
	for (int i = 0; i < size; i++) {
		if (rows[i] == col) {
			(* ret) = i;
		}
	}
}
